#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

//This is a little wrapper that checks for error codes returned by CUDA API calls
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void reduce_atomic(double *c, double *result, int n) {
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (tid < n)
        atomicAdd(result, c[tid]);
}

__global__ void reduce_shared1(double *c, double *result, int n) {
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ double local[1024];
 
    // Make sure we do not go out of bounds
    if (tid < n)
      local[threadIdx.x] = c[tid];
    else
      local[threadIdx.x] = 0.0;

    __syncthreads();
    if (threadIdx.x == 0) {
      double sum = 0.0;
      for (int i = 0; i < 1024; i++) sum += local[i];
      atomicAdd(result, sum);
    }
}

__global__ void reduce_shared2(double *c, double *result, int n) {
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ double local[1024];
 
    // Make sure we do not go out of bounds
    if (tid < n)
      local[threadIdx.x] = c[tid];
    else
      local[threadIdx.x] = 0.0;

    for (int d = blockDim.x >> 1; d >= 1; d >>= 1) {
      __syncthreads();
      if (threadIdx.x < d) local[threadIdx.x] += local[threadIdx.x+d];
    }

    if (threadIdx.x == 0) {
      atomicAdd(result, local[0]);
    }
}


// CUDA kernel. Each thread takes care of one element of c
__global__ void first(double *c, int n)
{
    // Get our global thread ID
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (tid < n)
        c[tid] = tid;
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 1<<30;
    //Host vector
    double *h_c;
    double h_result;
    //Device output vector
    double *d_c;
    double *d_result;
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
    // Allocate memory on host
    h_c = (double*)malloc(bytes);
    // Allocate memory on GPU
    // Note how we use the cudaCheck wrapper to check for error codes returned
    cudaCheck(hipMalloc(&d_c, bytes));
    cudaCheck(hipMalloc(&d_result, sizeof(double)));
    cudaCheck(hipMemset(d_result, 0, sizeof(double)));
    // Copy host vectors to device
    int blockSize, gridSize;
    // Number of threads in each thread block
    blockSize = 1024;
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
    // Execute the kernel
    first<<<gridSize, blockSize>>>(d_c, n);
    // Synchronize
    cudaCheck(hipDeviceSynchronize());
    auto t1 = std::chrono::high_resolution_clock::now();
    reduce_atomic<<<gridSize,blockSize>>>(d_c, d_result, n);
    cudaCheck(hipDeviceSynchronize());
    auto t2 = std::chrono::high_resolution_clock::now();
    std::cout << "took "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t2-t1).count()
              << " milliseconds\n";
    cudaCheck(hipMemset(d_result, 0, sizeof(double)));
    t1 = std::chrono::high_resolution_clock::now();
    reduce_shared1<<<gridSize,blockSize>>>(d_c, d_result, n);
    cudaCheck(hipDeviceSynchronize());
    t2 = std::chrono::high_resolution_clock::now();
    std::cout << "took "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t2-t1).count()
              << " milliseconds\n";

    cudaCheck(hipMemset(d_result, 0, sizeof(double)));
    t1 = std::chrono::high_resolution_clock::now();
    reduce_shared2<<<gridSize,blockSize>>>(d_c, d_result, n);
    cudaCheck(hipDeviceSynchronize());
    t2 = std::chrono::high_resolution_clock::now();
    std::cout << "took "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t2-t1).count()
              << " milliseconds\n";




    // Copy array back to host
    cudaCheck(hipMemcpy( &h_result, d_result, sizeof(double), hipMemcpyDeviceToHost ));
    // Print resulting array sequentially on the GPU
    printf("%g\n", h_result);
 
    // Release device memory
    hipFree(d_c);
    hipFree(d_result);
 
    // Release host memory
    free(h_c);
 
    return 0;
}

