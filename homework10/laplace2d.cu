#include "hip/hip_runtime.h"
/* Copyright (c) 2012, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <omp.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>


//This is a little wrapper that checks for error codes returned by CUDA API calls
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ static double my_atomic_max(double* address, double val) {
    unsigned long long int *address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    while (val > __longlong_as_double(old)) {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val));
        }
    return __longlong_as_double(old);
}

__global__ void my_stencil_kernel(double *A, double *Anew, int imax, int jmax, double *d_x,  double *error) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    __shared__ double local[6*14];
    if (i >= 1 && i < imax+1 && j >= 1 && j < jmax+1) {
        Anew[(j)*(imax+2)+i] = 0.25f * ( A[(j)*(imax+2)+i+1] + A[(j)*(imax+2)+i-1]
                            + A[(j-1)*(imax+2)+i] + A[(j+1)*(imax+2)+i]);
        d_x[(j)*(imax+2)+i] = fabs(Anew[(j)*(imax+2)+i]-A[(j)*(imax+2)+i]);
    }

    int tid = threadIdx.x+threadIdx.y*blockDim.x;

    if (i >= 1 && i < imax+1 && j >= 1 && j < jmax+1) 
      local[tid] = d_x[(j)*(imax+2)+i];
    else
      local[tid] = 0.0;

    if (tid == 0) {
      double local_error = 0.0;
      for (int i = 0; i < 6*14; i++) {
          if (local[i] > local_error) local_error = local[i];
      }
      my_atomic_max(error, local_error);
    }
}



__global__ void my_copy_kernel(double *A, double *Anew, int imax, int jmax) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= 1 && i < imax+1 && j >= 1 && j < jmax+1)
        A[(j)*(imax+2)+i] = Anew[(j)*(imax+2)+i];    
}

int main(int argc, char** argv)
{
    //Size along y
    int jmax = 4094;
    //Size along x
    int imax = 4094;
    int iter_max = 1000;
    
    const double pi  = 2.0 * asin(1.0);
    const double tol = 1.0e-5;
    double h_error = 1.0;

    double *A;
    double *Anew;
    double *y0;

    A    = (double *)malloc((imax+2) * (jmax+2) * sizeof(double));
    Anew = (double *)malloc((imax+2) * (jmax+2) * sizeof(double));
    y0   = (double *)malloc((imax+2) * sizeof(double));

    memset(A, 0, (imax+2) * (jmax+2) * sizeof(double));
    
    // set boundary conditions
    for (int i = 0; i < imax+2; i++)
      A[(0)*(imax+2)+i]   = 0.0;

    for (int i = 0; i < imax+2; i++)
      A[(jmax+1)*(imax+2)+i] = 0.0;
    
    for (int j = 0; j < jmax+2; j++)
    {
        y0[j] = sin(pi * j / (jmax+1));
        A[(j)*(imax+2)+0] = y0[j];
    }

    for (int j = 0; j < imax+2; j++)
    {
        y0[j] = sin(pi * j/ (jmax+1));
        A[(j)*(imax+2)+imax+1] = y0[j]*exp(-pi);
    }
    
    printf("Jacobi relaxation Calculation: %d x %d mesh\n", imax+2, jmax+2);
    
    double t1 = omp_get_wtime();
    int iter = 0;
    
    for (int i = 1; i < imax+2; i++)
       Anew[(0)*(imax+2)+i]   = 0.0;

    for (int i = 1; i < imax+2; i++)
       Anew[(jmax+1)*(imax+2)+i] = 0.0;

    for (int j = 1; j < jmax+2; j++)
        Anew[(j)*(imax+2)+0]   = y0[j];

    for (int j = 1; j < jmax+2; j++)
        Anew[(j)*(imax+2)+jmax+1] = y0[j]*expf(-pi);
    
    double *d_A;
    double *d_Anew;
    double *d_x;
    double *d_error;
    cudaCheck(hipMalloc(&d_A, (imax+2) * (jmax+2) * sizeof(double)));
    cudaCheck(hipMemcpy( d_A, A, (imax+2) * (jmax+2) * sizeof(double), hipMemcpyHostToDevice ));
    cudaCheck(hipMalloc(&d_Anew, (imax+2) * (jmax+2) * sizeof(double)));
    cudaCheck(hipMemcpy( d_Anew, Anew, (imax+2) * (jmax+2) * sizeof(double), hipMemcpyHostToDevice ));
    cudaCheck(hipMalloc(&d_x, (imax+2) * (jmax+2) * sizeof(double)));
    cudaCheck(hipMemset(d_x, 0, (imax+2) * (jmax+2) * sizeof(double)));
    cudaCheck(hipMalloc(&d_error, sizeof(double)));

    while ( h_error > tol && iter < iter_max )
    {
        cudaCheck(hipMemset(d_error, 0.0, sizeof(double)));

        dim3 block(16,4);
        dim3 grid((imax+2-1)/16+1, (jmax+2-1)/4+1);
        my_stencil_kernel<<<grid,block>>>(d_A, d_Anew, imax, jmax, d_x, d_error);
        // Synchronize
        cudaCheck(hipDeviceSynchronize());
        cudaCheck(hipMemcpy( &h_error, d_error, sizeof(double), hipMemcpyDeviceToHost ));

        //No stencil accesses to Anew, no halo exchange necessary
        my_copy_kernel<<<grid,block>>>(d_A, d_Anew, imax, jmax);
        if(iter % 100 == 0) printf("%5d, %0.6f\n", iter, h_error);
        
        iter++;
    }

    double runtime = omp_get_wtime()-t1;
 
    printf(" total: %f s\n", runtime);
}

